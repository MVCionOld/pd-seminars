
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    hipError_t cuda_stat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int array_size = (1 << 22);
    float* h_x = new float[array_size];

    for (int i = 0; i < array_size; ++i) {
        h_x[i] = i * 2.0f;
    }

    float* d_x;

    cuda_stat = hipMalloc(&d_x, sizeof(float) * array_size);

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(
        array_size,
        sizeof(*h_x),
        h_x,
        /* space by host */ 1,
        d_x,
        /* space by device */ 1
    );
    int result;

    stat = hipblasIsamax(handle, array_size / 3, d_x, 3, &result);
    
    int index = (result - 1) * 3;

    std::cout << index << " " << h_x[index] << std::endl;
    // 0, 3, 6, 9, ...
    // 1, 2, 3, 4, 5, ...
    // max element located at (1398102 - 1) * 3

    std::cout << result << std::endl;

    return 0;
}
