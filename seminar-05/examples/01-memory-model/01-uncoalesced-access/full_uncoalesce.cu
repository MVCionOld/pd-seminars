
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>

#define ILP 8

__global__
void add(int n, float* x, float* y, float* z) {
    int tid = threadIdx.x + ILP * blockDim.x * blockIdx.x;
    for (int i = 0; i < ILP; ++i) {
        int current_tid = tid + i * blockDim.x;
        
        z[current_tid] = 2.0f * x[current_tid] + y[current_tid];
    }
}

__global__
void stupid_add(int n, float* x, float* y, float* z) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int actual_tid = ILP * index;
    for (int i = 0; i < ILP; ++i) {
        int current_tid = actual_tid + i;
        z[current_tid] = 2.0f * x[current_tid] + y[current_tid];
    }
}


int main() {
	int N = 1 << 28;
	size_t size = N * sizeof(float);
	float *x = (float*)malloc(size);
	float *y = (float*)malloc(size);
    float *z = (float*)malloc(size);

	float *d_x, *d_y, *d_z;

	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);


	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}


	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	int blockSize = 256;

	int numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);
	add<<<numBlocks / ILP, blockSize>>>(N, d_x, d_y, d_z);
    hipEventRecord(stop);

	hipMemcpy(z, d_z, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    hipEvent_t start1;
    hipEvent_t stop1;
    
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    hipEventRecord(start1);
    stupid_add<<<numBlocks / ILP, blockSize>>>(N, d_x, d_y, d_z);
    hipEventRecord(stop1);

    hipMemcpy(z, d_z, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop1);
    
    float milliseconds = 0;
    float milliseconds_stupid = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventElapsedTime(&milliseconds_stupid, start1, stop1);
    std::cout << milliseconds << " elapsed normal" << std::endl;
    std::cout << milliseconds_stupid << " elapsed stupid" << std::endl;

	hipFree(d_x);
	hipFree(d_y);
    hipFree(d_z);
	free(x);
	free(y);
    free(z);
	return 0;
}
