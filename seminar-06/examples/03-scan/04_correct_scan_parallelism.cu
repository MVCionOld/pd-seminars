
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>


__global__ void Scan(int* in_data, int* out_data) {
    // in_data ->  [1 2 3 4 5 6 7 8], block_size 4
    // block_idx -> [0 0 0 0 1 1 1 1 ]
    
    extern __shared__ int shared_data[];
    // block_idx = 0

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = in_data[index];

    // shared_data -> [1, 2, 3, 4]
    __syncthreads();
    
    // shift = 2^(d - 1)
    for (unsigned int shift = 1; shift < blockDim.x; shift <<= 1 ) {
        int ai = shift * (2 * tid + 1) - 1; 
        // tid = 0, shift = 1, ai = 0; bi = 1; 
        // tid = 16, shift = 1, ai = 32 = 0; bi = 33 = 1;
        int bi = shift * (2 * tid + 2) - 1;

        if (bi < blockDim.x) {
            shared_data[bi] += shared_data[ai];
        }

        __syncthreads();
    }

    if (tid == blockDim.x  - 1) {
        shared_data[tid] = 0;
    }

    __syncthreads();

    int temp;
    for (unsigned int shift = blockDim.x / 2; shift > 0; shift >>= 1) {
        int i = 2 * shift * (tid  + 1) - 1;
        if (i < blockDim.x) {
            temp = shared_data[i - shift]; // blue in temp

            // temp = 4
            shared_data[i - shift] = shared_data[i]; // orange

            // 1 2 1 0 1 2 1 0 // temp = 4
            shared_data[i] = temp + shared_data[i];
        }
        __syncthreads();

        // step 2
        // 1 2 1 0 1 2 1 4
        // tid == 3
        // temp = 2
        // 1 0 1 0 1 2 1 4
        // 1 0 1 2 1 2 1 4
        // tid == 7
        // temp = 2
        // 1 0 1 2 1 4 1 4
        // 1 0 1 2 1 4 1 6

        // tid == 1
        // temp = 0
        // 0 1 1 2 1 4 1 6
        // 0 1 1 2 1 4 1 6
        // tid == 3
        // temp = 1
        // 0 1 2 2 1 4 1 6
        // 0 1 2 3 1 4 1 6
        // tid == 5
        // temp = 1
        // 0 1 2 3 4 4 1 6
        // 0 1 2 3 4 5 1 6
        // tid == 7
        // temp = 1
        // 0 1 2 3 4 5 6 6
        // 0 1 2 3 4 5 6 7
    }
    // if (blockIdx.x == 16383) {
    //     printf("%d %d %d\n", tid, shared_data[tid], index);
    //     // std::cout << shared_data[tid] << std::endl;
    // }
    // block_idx = 0 -> [a0, a1, a2, a3]
    // block_idx = 1 -> [a4, a5, a6, a7]
    out_data[index] = shared_data[tid];

    __syncthreads();

    // out_data[block_idx == 0] = [1, 3, 6, 10]

    // out_data[block_idx == 1] = [5, 11, 18, 26]

}


int main() {
    const int block_size = 256;

    const int array_size = 1 << 22;
    int* h_array = new int[array_size];
    for (int i = 0; i < array_size; ++i) {
        h_array[i] = 1;
    }

    // int* output = new int[array_size];

    int* d_array;
    hipMalloc(&d_array, sizeof(int) * array_size);

    hipMemcpy(d_array, h_array, sizeof(int) * array_size, hipMemcpyHostToDevice);


    int num_blocks = array_size / block_size;

    int* d_localscan;
    hipMalloc(&d_localscan, sizeof(int) * array_size);
    int* h_localscan = new int[array_size];

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);
    Scan<<<num_blocks, block_size, sizeof(int) * block_size>>>(d_array, d_localscan);


    hipEventRecord(stop);

    hipMemcpy(h_localscan, d_localscan, sizeof(int) * array_size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << " elapsed" << std::endl;

    std::cout << h_localscan[array_size - 1] << std::endl;

    delete[] h_array;
    delete[] h_localscan;


}