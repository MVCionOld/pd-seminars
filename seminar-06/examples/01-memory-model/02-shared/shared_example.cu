
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cassert>


#define BLOCKSIZE 512

__global__ void ComputeThreeSum(int n, int* input, int* result) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int local_tid = threadIdx.x;
    __shared__ int s_data[BLOCKSIZE]; // unique for every block!

    int tmp = 0;
    s_data[local_tid] = input[tid]; // copy data to shared memory
    
    __syncthreads();

    if (local_tid > 0) {
        tmp = s_data[local_tid - 1];
    } else if (tid > 0) {
        tmp = input[tid - 1];
    }

    if (local_tid + 1 < BLOCKSIZE) {
        tmp = tmp + s_data[local_tid + 1];
    } else if (tid + 1 < n) {
        tmp = tmp + input[tid + 1];    
    }

    tmp = tmp + s_data[local_tid];
    result[tid] = tmp;
}


int main() {
    int N = 1 << 28;

    int* h_array = new int[N];
    int* h_diff = new int[N];
    for (int i = 0; i < N; ++i) {
        h_array[i] = 1;
    }
    
    int* d_array;
    int* d_diff;
    unsigned int size = N * sizeof(int);
    hipMalloc(&d_array, size);
    hipMalloc(&d_diff, size);

    hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);
    
    int num_blocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    ComputeThreeSum<<<num_blocks, BLOCKSIZE>>>(N, d_array, d_diff);
    hipEventRecord(stop);


    hipMemcpy(h_diff, d_diff, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds;


    hipEventElapsedTime(&milliseconds, start, stop);

    for (int i = 1; i < N - 1; ++i) {
        if (h_diff[i] != 3) {
            std::cout << i << " " << h_diff[i] << std::endl;    
        }
        assert(h_diff[i] == 3);
    }

    std::cout << milliseconds << " elapsed" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_array);
    hipFree(d_diff);
    delete[] h_array;
    delete[] h_diff;

}
