
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cassert>


#define BLOCKSIZE 512

__global__ void ComputeTriSum(int n, int* input, int* result) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int result_tmp = 0;

    if (tid > 0) {
        result_tmp = input[tid - 1];
    }
    if (tid + 1 < n) {
        result_tmp = result_tmp + input[tid + 1];    
    }

    result_tmp = result_tmp + input[tid];
    result[tid] = result_tmp;
}


int main() {
    int N = 1 << 28;

    int* h_array = new int[N];
    int* h_diff = new int[N];
    for (int i = 0; i < N; ++i) {
        h_array[i] = 1;
    }
    
    int* d_array;
    int* d_diff;
    unsigned int size = N * sizeof(int);
    hipMalloc(&d_array, size);
    hipMalloc(&d_diff, size);

    hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);
    
    int num_blocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    ComputeTriSum<<<num_blocks, BLOCKSIZE>>>(N, d_array, d_diff);
    hipEventRecord(stop);


    hipMemcpy(h_diff, d_diff, size, hipMemcpyDeviceToHost);

    float milliseconds;
    hipEventSynchronize(stop);


    hipEventElapsedTime(&milliseconds, start, stop);

    for (int i = 1; i < N - 1; ++i) {
        if (h_diff[i] != 3) {
            std::cout << i << " " << h_diff[i] << std::endl;    
        }
        assert(h_diff[i] == 3);
    }

    std::cout << milliseconds << " elapsed" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_array);
    hipFree(d_diff);
    delete[] h_array;
    delete[] h_diff;

}
