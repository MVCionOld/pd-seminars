
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKSIZE 256

__global__ void StupidSumArray(int* array, int* result) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    result[tid] = 0;
    for (int i = tid * 1024; i < (tid + 1) * 1024; ++i) {
        result[tid] += array[i];
    }
}


int main() {
    int N = 1 << 18;
    int *h_x = new int[N];

    for (int i = 0; i < N; ++i) {
        h_x[i] = 1;
    }
    int *d_x;
    int size = sizeof(int) * N;
    hipMalloc(&d_x, size);

    int* h_result = new int[256];
    for (int i = 0; i < BLOCKSIZE; ++i) {
        h_result[i] = 0;
    }
    int *d_result;
    hipMalloc(&d_result, sizeof(int) * 256); 

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, h_result, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    StupidSumArray<<<1, 256>>>(d_x, d_result);

    hipEventRecord(stop);

    hipMemcpy(h_result, d_result, sizeof(int) * 256, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    for (int i = 0; i < 256; ++i) {
        std::cout << i << " " << h_result[i] << std::endl;
    }

    std::cout << ms << std::endl;
    hipFree(d_x);
    hipFree(d_result);
    delete[] h_result;
    delete[] h_x;

}
