
#include <hip/hip_runtime.h>
#include <iostream>

__device__ void WarpReduce(volatile int* shared_data, int tid) {
    shared_data[tid] += shared_data[tid + 32];
    shared_data[tid] += shared_data[tid + 16];
    shared_data[tid] += shared_data[tid + 8];
    shared_data[tid] += shared_data[tid + 4];
    shared_data[tid] += shared_data[tid + 2];
    shared_data[tid] += shared_data[tid + 1];
}

__global__ void Reduce(int* in_data, int* out_data) {
    extern __shared__ int shared_data[];

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    shared_data[tid] = in_data[index] + in_data[index + blockDim.x];
    __syncthreads();
    
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    if (tid < 32) {
        WarpReduce(shared_data, tid);
    }
    
    if (tid == 0) {
        out_data[blockIdx.x] = shared_data[0];
    }
}


int main() {
    const int block_size = 256;
    // __shared__ int shared_data[];

    const int array_size = 1 << 22;
    int* h_array = new int[array_size];
    for (int i = 0; i < array_size; ++i) {
        h_array[i] = 1;
    }

    int* d_array;
    hipMalloc(&d_array, sizeof(int) * array_size);

    hipMemcpy(d_array, h_array, sizeof(int) * array_size, hipMemcpyHostToDevice);

    int num_blocks = array_size / block_size / 2;

    int* d_blocksum;
    hipMalloc(&d_blocksum, sizeof(int) * num_blocks);
    int* h_blocksum = new int[num_blocks];

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);

    Reduce<<<num_blocks, block_size, sizeof(int) * block_size>>>(d_array, d_blocksum);

    hipEventRecord(stop);

    hipMemcpy(h_blocksum, d_blocksum, sizeof(int) * num_blocks, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << " elapsed" << std::endl;
    
    int sum = 0;
    for (int i = 0; i < num_blocks; ++i) {
        sum += h_blocksum[i];
    }

    std::cout << sum << std::endl;

    hipFree(d_blocksum);
    hipFree(d_array);
    delete[] h_array;
    delete[] h_blocksum;

}