
#include <hip/hip_runtime.h>
#include <iostream>

/*
 Та же суть, что и в предыдущем примере, но при загрузке
 в shared memory мы кладем преподсчитанные частичне суммы
 */
__global__ void Reduce(int* in_data, int* out_data) {
    extern __shared__ int shared_data[];

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    shared_data[tid] = in_data[index] + in_data[index + blockDim.x] + in_data[index + blockDim.x * 2] + in_data[index + blockDim.x * 3];
    __syncthreads();
    
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out_data[blockIdx.x] = shared_data[0];
    }
}


int main() {
    const int block_size = 256;
    // __shared__ int shared_data[];

    const int array_size = 1 << 22;
    int* h_array = new int[array_size];
    for (int i = 0; i < array_size; ++i) {
        h_array[i] = 1;
    }

    int* d_array;
    hipMalloc(&d_array, sizeof(int) * array_size);

    hipMemcpy(d_array, h_array, sizeof(int) * array_size, hipMemcpyHostToDevice);

    int num_blocks = array_size / block_size / 4;

    int* d_blocksum;
    hipMalloc(&d_blocksum, sizeof(int) * num_blocks);
    int* h_blocksum = new int[num_blocks];

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);

    Reduce<<<num_blocks, block_size, sizeof(int) * block_size>>>(d_array, d_blocksum);

    hipEventRecord(stop);

    hipMemcpy(h_blocksum, d_blocksum, sizeof(int) * num_blocks, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << " elapsed" << std::endl;
    
    int sum = 0;
    for (int i = 0; i < num_blocks; ++i) {
        sum += h_blocksum[i];
    }

    std::cout << sum << std::endl;

    hipFree(d_blocksum);
    hipFree(d_array);
    delete[] h_array;
    delete[] h_blocksum;

}
