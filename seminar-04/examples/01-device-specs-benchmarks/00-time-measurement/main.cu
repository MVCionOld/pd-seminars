
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__
void add(int n, float* x, float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n; i += stride) {
		y[i] = x[i] + y[i];
	}	
}


int main() {
	int N = 1 << 28;
	size_t size = N * sizeof(float);
	float *x = (float*)malloc(size);
	float *y = (float*)malloc(size);

	float *d_x, *d_y;

	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);


	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}


	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	int blockSize = 256;

	int numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);
	add<<<numBlocks, blockSize>>>(N, d_x, d_y);

    // cudaEventRecord(stop);

	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << " elapsed" << std::endl;

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
	return 0;
}
