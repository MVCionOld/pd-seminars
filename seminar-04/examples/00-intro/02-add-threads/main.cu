
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__
void add(int n, float* x, float* y) {
	int index = threadIdx.x;
	int stride = blockDim.x;

	for (int i = index; i < n; i += stride) {
		y[i] = x[i] + y[i];
	}	
}


int main() {
	int N = 1 << 28;
	float *x, *y;

	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));


	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	add<<<1, 256>>>(N, x, y);

	hipDeviceSynchronize();	
	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	hipFree(x);
	hipFree(y);
	return 0;
}
