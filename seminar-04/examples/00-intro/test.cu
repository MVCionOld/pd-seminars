
#include <hip/hip_runtime.h>
#include<cstdio>

using namespace std;

__global__ void add(const int *a, const int *b, int *c)
{
	int i = threadIdx.x;
	c[i] = a[i] * *b;
}

int main(void)
{
	int count = 100;
	int size = sizeof(int) * count;
	int *cpu_a = (int *)malloc(size);	int *gpu_a; hipMalloc((void**)&gpu_a, size);
	int  cpu_b = 5;						int *gpu_b; hipMalloc((void**)&gpu_b, sizeof(int));
	int *cpu_c = (int *)malloc(size);	int *gpu_c; hipMalloc((void**)&gpu_c, size);

	for(int i=0; i<count; i++) cpu_a[i]=i;

	hipMemcpy(gpu_a,  cpu_a, size,			hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, &cpu_b, sizeof(int),	hipMemcpyHostToDevice);

	add<<<1, count>>>(gpu_a, gpu_b, gpu_c);

	hipMemcpy(cpu_c, gpu_c, size, hipMemcpyDeviceToHost);

	for(int i=0; i<count; i++)
		printf("%d * %d = %dn", cpu_a[i], cpu_b, cpu_c[i]);
	free(cpu_a);	hipFree(gpu_a);
					hipFree(gpu_b);
	free(cpu_c);	hipFree(gpu_c);
}

