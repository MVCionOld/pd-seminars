
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 256


void FillMatrix(float* matrix, int height, int width) {
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			if (i == j) {
				matrix[i * width + j] = 1;
			} else {
				matrix[i * width + j] = 0;
			}
		}
	}
}

void PrintMatrix(float *matrix, int height, int width) {

	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			std::cout << i << " " << j << " " << matrix[i * width + j] << "\n";
		}
	}
}


__global__
void MatrixMul(float* A, float* B, float* C, int mid_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int height = blockDim.x * gridDim.x;
    int width = blockDim.y * gridDim.y;

    C[i * width + j] = .0f;

    for (int k = 0; k < mid_size; ++k) {
        C[i * width + j] += A[i * mid_size + k] * B[k * width + j];
    }
}


int main() {

	float *h_A;
	float *h_B;
	float *h_C;
	// h_A 128 * 384,
	// h_B 384 * 256
	// h_C 128 * 256

	h_A = new float[128 * 384];
	h_B = new float[384 * 256];
	h_C = new float[128 * 256];

	FillMatrix(h_A, 128, 384);
	FillMatrix(h_B, 384, 256);

    // PrintMatrix(h_A, 128, 384);


	float* d_A;
	float* d_B;
	float* d_C;

	hipMalloc(&d_A, sizeof(float) * 128 * 384);
	hipMalloc(&d_B, sizeof(float) * 384 * 256);
	hipMalloc(&d_C, sizeof(float) * 128 * 256);

    hipMemcpy(d_A, h_A, sizeof(float) * 128 * 384, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * 384 * 256, hipMemcpyHostToDevice);

    // kernel call
    dim3 num_blocks(8, 16);
    dim3 block_size(16, 16);

    MatrixMul<<<num_blocks, block_size>>>(d_A, d_B, d_C, 384);

    hipMemcpy(h_C, d_C, sizeof(float) * 128 * 256, hipMemcpyDeviceToHost);
    PrintMatrix(h_C, 128, 256);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	return 0;
}
